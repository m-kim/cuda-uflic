#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <type_traits>

#include "Draw.cuh"
#include "Sharpen.cuh"
#include "Jitter.cuh"
#include "Evaluator.cuh"
#include "Integrator.cuh"
#include "UFLIC.cuh"
#include "Reader.h"

template<class T>
struct normale {
	__host__ __device__ T operator()(const T &x, const T &y) const {
		T reval = 0;
		if (x > 0)
			reval = x / y;

		return reval;
	}
};


struct resetParticles {
	resetParticles(uint2 _d) { dim = _d; }
	__host__ __device__ float2 operator()(const uint &idx) {
		 uint y = idx / dim.x;
		 uint x = idx % dim.x;
		 return make_float2(x + 0.5, y + 0.5);

	}
	
	uint2 dim;
};

struct prg
{
	unsigned char a, b;

	__host__ __device__
		prg(unsigned char _a = 0.f, unsigned char _b = 1.f) : a(_a), b(_b) {};

	__host__ __device__
		float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<unsigned char> dist(a, b);
		rng.discard(n);

		return dist(rng);
	}
};

template<typename VecComponentType>
void saveAs(std::string fileName, 
	thrust::host_vector<VecComponentType> canvas, 
	size_t Width, size_t Height) {
	std::ofstream of(fileName.c_str(), std::ios_base::binary | std::ios_base::out);
	of << "P6" << std::endl << Width << " " << Height << std::endl << 255 << std::endl;
	//ColorBufferType::PortalConstControl colorPortal = this->ColorBuffer.GetPortalConstControl();
	for (size_t yIndex = Height - 1; yIndex >= 0; yIndex--)
	{
		for (size_t xIndex = 0; xIndex < Width; xIndex++)
		{
			VecComponentType val = canvas[yIndex * Width + xIndex];
			
			uint4 tuple = make_uint4(val, val, val, val);
			of << (unsigned char)(tuple.x);
			of << (unsigned char)(tuple.y);
			of << (unsigned char)(tuple.z);
		}
	}
	of.close();
}


int main(int argc, char **argv)
{
	const size_t ttl = 4, loop_cnt = 12;
	typedef float VecType;
	typedef unsigned char FieldType;

  typedef float2 VecField;


  thrust::host_vector<VecField> vecs;
  thrust::device_vector<VecField> d_vecs;
  //std::shared_ptr<Reader<VecType, Size, ReaderVTK<VecType, Size>>> reader(new ReaderVTK<VecType, Size>("BField_2d.vtk"));
  //std::shared_ptr<Reader<VecType, Size,  ReaderPS<VecType, Size,ReaderXGC<VecType,Size>>>> reader(new ReaderPS<VecType, Size, ReaderXGC<VecType,Size>>("/home/mkim/vtkm-uflic/psi2q/2D_packed/psi2D_packed_normalized_256_99.vec", vtkm::Id2(256,256), Bounds(0,256,0,256)));
  //std::shared_ptr<ReaderPS<VecType, Size, ReaderXGC<VecType, Size>>> reader(new ReaderXGC<VecType, Size>("/home/mkim/vtkm-uflic/psi2q/2D_packed/psi2D_packed_512_", vtkm::Id2(512,512), Bounds(0,512,0,512), loop_cnt));
  //std::shared_ptr<ReaderPS<VecType, Size, ReaderXGC<VecType, Size>>> reader(new ReaderXGC<VecType, Size>("XGC_", vtkm::Id2(96,256), Bounds(0,96,0,256)));
  //typedef VectorField<VecType,Size> EvalType;


  int x = 512;
  int y = 256;
  if (argc > 1){
    x = atoi(argv[1]);
    y = atoi(argv[2]);
  }
  std::shared_ptr<Reader<VecType,ReaderCalc<VecType>>> reader(new ReaderCalc<VecType>("XGC_", make_uint2(x,y), make_float2(0,0),make_float2(x,y)));
  typedef DoubleGyreField<VecType> EvalType;



  typedef RK4Integrator<EvalType, VecType> IntegratorType;

  reader->read(vecs);

  auto t0 = std::chrono::high_resolution_clock::now();

  uint2 dim = { 256,256 };

  //vtkm::Id2 dim = reader->dim;
  float2 spacing = reader->spacing;
  //Bounds bounds = reader->bounds;

	std::vector<thrust::host_vector<VecField>> h_l(ttl), h_r(ttl);

  for (int i = 0; i < ttl; i++) {
    for (int y = 0; y<dim.y; y++) {
      for (int x = 0; x<dim.x; x++) {
        h_l[i].push_back(make_float2(x + 0.5, y + 0.5));
        h_r[i].push_back(make_float2(x + 0.5, y + 0.5));
      }
    }
  }
  std::vector<thrust::device_vector<VecField>> d_l(ttl), d_r(ttl);
	for (int i = 0; i<ttl; i++) {
		d_l[i] = h_l[i];
		d_r[i] = h_r[i];
	}

	
	//vecArray = vtkm::cont::make_ArrayHandle(&vecs[0], vecs.size());


	thrust::host_vector<FieldType> h_canvas[ttl], h_propertyField[2], h_omega(dim.x * dim.y, 0), h_tex(dim.x * dim.y, 0);
	VecType t = 0;
	const VecType dt = 0.1;
		for (int i = 0; i < 2; i++) {
		h_propertyField[i].resize(dim.x * dim.x, 0);
	}

	for (int i = 0; i < ttl; i++) {
		h_canvas[i].resize(dim.x * dim.y, 0);
	}
	for (int i = 0; i < h_canvas[0].size(); i++) {
		h_tex[i] = h_canvas[0][i] = rand() % 255;
	}

	thrust::device_vector<FieldType > d_canvas[ttl], d_propField[2], d_omega, d_tex;
	for (int i = 0; i < ttl; i++) {
		d_canvas[i] = h_canvas[i];
	}
	d_propField[0] = h_propertyField[0];
	d_propField[1] = h_propertyField[1];
	d_omega = h_omega;
	d_tex = h_tex;

  //DrawLineWorkletType drawline(bounds, dim);
	//DoSharpen<FieldType, DeviceAdapter> dosharp(dim);
	//DoJitter<FieldType, DeviceAdapter> dojitter(dim);
  //vtkm::cont::ArrayHandleCounting<vtkm::Id> indexArray(vtkm::Id(0), 1, propFieldArray[0].GetNumberOfValues());
	thrust::counting_iterator<uint> indexArray_begin(0), indexArray_end;
	indexArray_end = indexArray_begin + (dim.x * dim.y);

  for (int loop = 0; loop < loop_cnt; loop++) {
	EvalType eval(t, make_float2(0,0), make_float2(dim.x, dim.y), spacing);
    IntegratorType integrator(eval, 3.0);
    //ParticleAdvectionWorkletType advect(integrator);
    //std::cout << "t: " << t << std::endl;

	//vtkm::worklet::DispatcherMapField<ResetParticles<VecType,Size>> resetDispatcher(dim[0]);
    //resetDispatcher.Invoke(indexArray, sl[loop%ttl]);
	thrust::transform(indexArray_begin, indexArray_end, d_l[loop%ttl].begin(), resetParticles(dim));
		//reset the current canvas
		for (int i = 0; i < d_canvas[loop % ttl].size(); i++) {
			d_canvas[loop % ttl][i] = rand() % 255;
		}

    thrust::fill(d_propField[0].begin(), d_propField[0].end(), 0);
	thrust::fill(d_propField[1].begin(), d_propField[1].end(), 0);
	thrust::fill(d_omega.begin(), d_omega.end(), 0);

		for (int i = 0; i < min(ttl, static_cast<size_t>(loop)+1); i++) {
			//advect.Run(sl[i], sr[i], vecArray);
			
			advect<IntegratorType, VecField, VecField> << <dim.x*dim.y/32, 32 >> > (
				thrust::raw_pointer_cast(d_l[i].data()),
				thrust::raw_pointer_cast(vecs.data()),
				integrator,
				thrust::raw_pointer_cast(d_r[i].data())
				);
			//drawline.Run(canvasArray[i], propFieldArray[0], omegaArray, sl[i], sr[i]);
			drawline<<<dim.x*dim.y/32, 32>>>(thrust::raw_pointer_cast(d_canvas[i].data()),
				thrust::raw_pointer_cast(d_omega.data()),
				thrust::raw_pointer_cast(d_l[i].data()),
				thrust::raw_pointer_cast(d_r[i].data()),
				thrust::raw_pointer_cast(d_propField[0].data())
				);


		}

		//sr.swap(sl);
		d_r.swap(d_l);

		//donorm.Run(propFieldArray[0], omegaArray, propFieldArray[1]);
		thrust::transform(d_propField[0].begin(), d_propField[0].end(), d_omega.begin(), d_propField[1].begin(), normale<unsigned char>());
		
		h_propertyField[1] = d_propField[1];
    std::stringstream fn;
    fn << "uflic-" << loop << ".pnm";
    saveAs(fn.str().c_str(), h_propertyField[1], dim.x, dim.y);

    //REUSE omegaArray as a temporary cache to sharpen
    //dosharp.Run(propFieldArray[1], omegaArray);
	dim3 dimBlock(16, 16);
	dim3 dimGrid;
	dimGrid.x = (dim.x + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (dim.y + dimBlock.y - 1) / dimBlock.y;
	sharpen<unsigned char><<<dimGrid, dimBlock>>>(
		thrust::raw_pointer_cast(d_propField[1].data()),
		thrust::raw_pointer_cast(d_omega.data())
		);
    //dojitter.Run(omegaArray, texArray, canvasArray[(loop) % ttl]);

	auto data_tex_begin = thrust::make_zip_iterator(
		make_tuple(indexArray_begin, d_omega.begin(), d_tex.begin()));
	auto data_tex_end = thrust::make_zip_iterator(
		make_tuple(indexArray_end, d_omega.end(), d_tex.end()));
	thrust::transform(
		data_tex_begin,
		data_tex_end,
		d_canvas[(loop) % ttl].begin(),
		Jitter<FieldType>(dim, 256, 256 * 0.1, 256 * 0.9));


    t += dt;// / (vtkm::Float32)ttl + 1.0 / (vtkm::Float32)ttl;
    reader->next(vecs);
    //vecArray = vtkm::cont::make_ArrayHandle(&vecs[0], vecs.size());

	}

  auto t1 = std::chrono::high_resolution_clock::now();

  std::cout << "Finished dt: " << dt << " cnt: " << loop_cnt << " time: " << std::chrono::duration<double>(t1-t0).count() << "s" << std::endl;
    std::stringstream fn;
    fn << "uflic-final" << ".pnm";
	h_propertyField[1] = d_propField[1];
    saveAs(fn.str().c_str(), h_propertyField[1], dim.x, dim.y);


	//vtkm::rendering::Mapper mapper;
	//vtkm::rendering::Canvas canvas(512, 512);
	//vtkm::rendering::Scene scene;

	//scene.AddActor(vtkm::rendering::Actor(
	//	ds.GetCellSet(), ds.GetCoordinateSystem(), ds.GetField(fieldNm), colorTable));
	//vtkm::rendering::Camera camera;
	//SetCamera<ViewType>(camera, ds.GetCoordinateSystem().GetBounds());

	//vtkm::rendering::View2D view;
}
